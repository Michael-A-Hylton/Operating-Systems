#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

#define EPSILON (0.001f)
#define notEqual(x,y)     (fabs((x) - (y)) > EPSILON)

typedef unsigned long long SysClock;

static SysClock currentTicks() {
    timespec spec;
    clock_gettime(CLOCK_THREAD_CPUTIME_ID, &spec);
    return (SysClock)((float)(spec.tv_sec) * 1e9 + (float)(spec.tv_nsec));
}

static double secondsPerTick() {
    static int initialized = 0;
    static double secondsPerTick_val;
    if (initialized) return secondsPerTick_val;
    FILE *fp = fopen("/proc/cpuinfo","r");
    char input[1024];
    if (!fp) {
        fprintf(stderr, "resetScale failed: couldn't find /proc/cpuinfo.");
	exit(-1);
    }
    secondsPerTick_val = 1e-9;
    while (!feof(fp) && fgets(input, 1024, fp)) {
        float GHz, MHz;
	if (strstr(input, "model name")) {
	    char* at_sign = strstr(input, "@");
	    if (at_sign) {
	        char* after_at = at_sign + 1;
		char* GHz_str = strstr(after_at, "GHz");
		char* MHz_str = strstr(after_at, "MHz");
		if (GHz_str) {
		    *GHz_str = '\0';
		    if (1 == sscanf(after_at, "%f", &GHz)) {
		        //printf("GHz = %f\n", GHz);
			secondsPerTick_val = 1e-9f / GHz;
			break;
		    }
		} else if (MHz_str) {
		    *MHz_str = '\0';
		    if (1 == sscanf(after_at, "%f", &MHz)) {
		        //printf("MHz = %f\n", MHz);
			secondsPerTick_val = 1e-6f / GHz;
			break;
		    }
		}
	    }
	} else if (1 == sscanf(input, "cpu MHz : %f", &MHz)) {
	    //printf("MHz = %f\n", MHz);
	    secondsPerTick_val = 1e-6f / MHz;
	    break;
	}
    }
    fclose(fp);
    initialized = 1;
    return secondsPerTick_val;
}

static double currentSeconds() {
    return currentTicks() * secondsPerTick();
}

float toBW(int bytes, float sec) {
  return (float)(bytes) / (1024. * 1024. * 1024.) / sec;
}

__global__ void faxpy_1blk_kernel(int N, float alpha, float *x, float *y, float *result) {
    
    
    for(int i=0; i<N; i+=1){
    	result[i]=alpha*x[i]+y[i];
    }
    
}

__global__ void faxpy_mblk_kernel(int N, float alpha, float* x, float* y, float* result) {

    
    int tid=blockIdx.x * blockDim.x + threadIdx.x;
    if (tid<N){
    	result[tid]=alpha*x[tid]+y[tid];
    }
    
}

void faxpyCuda(int N, float alpha, float* xarray, float* yarray, float* resultarray) {
    printf("faxpyCuda");
    int totalBytes = sizeof(float) * 3 * N;

    // compute number of blocks and threads per block
    const int threadsPerBlock = 512;
    const int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

    float* device_x;
    float* device_y;
    float* device_result;

    device_x=(float *)malloc(sizeof(float)*N);
    device_y=(float *)malloc(sizeof(float)*N);
    device_result=(float *)malloc(sizeof(float)*N);
    float *d_device_x, *d_device_y, *d_device_result;
    hipMalloc((void**)&d_device_x, sizeof(float) * N);//
    hipMalloc((void**)&d_device_y, sizeof(float) * N);//
    hipMalloc((void**)&d_device_result, sizeof(float) * N);//


    // start timing after allocation of device memory
    double startTime = currentSeconds();

    //
    hipMemcpy(d_device_x, xarray, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_device_y, yarray, sizeof(float) * N, hipMemcpyHostToDevice);
    //
    
    
    double midTime1 = currentSeconds();

    faxpy_1blk_kernel<<<1,threadsPerBlock>>>(N, alpha, d_device_x, d_device_y, d_device_result); //
    
    faxpy_mblk_kernel<<<blocks,threadsPerBlock>>>(N, alpha, d_device_x, d_device_y, d_device_result) ;
    
    // IMPORTANT, wait for the completion at GPU
    hipDeviceSynchronize();

    double midTime2 = currentSeconds();

    //
    hipMemcpy(resultarray, d_device_result, sizeof(float) * N, hipMemcpyDeviceToHost);
    

    // end timing after result has been copied back into host memory
    double endTime = currentSeconds();

    hipError_t errCode = hipPeekAtLastError();
    if (errCode != hipSuccess) {
        fprintf(stderr, "WARNING: A CUDA error occured: code=%d, %s\n", errCode, hipGetErrorString(errCode));
    }

    double overallDuration = endTime - startTime;
    printf("Overall: %.3f ms\t\t[%.3f GB/s]\n", 1000.f * overallDuration, toBW(totalBytes, overallDuration));

    double transferDur = midTime1 - startTime;
    printf("xy array --> device %.3f ms\n", 1000.f * transferDur);

    double gpu_compute_dur = midTime2 - midTime1;
    printf("GPU computation duration %.3f ms\n", 1000.f * gpu_compute_dur);

    hipFree(d_device_x);
    hipFree(d_device_y);
    hipFree(d_device_result);
    free(device_x);
    free(device_y);
    free(device_result);
    

}

void faxpyCPU(int N, float alpha, float *xarray, float *yarray, float *resultarray) {
    double startTime = currentSeconds();
    for (int i = 0; i < N; i++) {
        resultarray[i] = alpha * xarray[i] + yarray[i];
    }
    double endTime = currentSeconds();
    double cpu_dur = endTime - startTime;
    printf("CPU computation duration %.3f ms\n", 1000.f * cpu_dur);
}

int main(int argc, char** argv)
{
    printf("main");
    int N = 20 * 1000 * 1000;

    const float alpha = 5.0f;
    const float max = 999.0f;
    float* xarray = (float *)malloc(sizeof(float)*N);
    float* yarray = (float *)malloc(sizeof(float)*N);
    float* resultarray = (float *)malloc(sizeof(float)*N);
    float* checkarray = (float *)malloc(sizeof(float)*N);

    for (int i=0; i<N; i++) {
	xarray[i] = ((float)rand()/(float)(RAND_MAX)) * max;
	yarray[i] = ((float)rand()/(float)(RAND_MAX)) * max;
        resultarray[i] = 0.f;
    }

    faxpyCuda(N, alpha, xarray, yarray, resultarray);

    faxpyCPU(N, alpha, xarray, yarray, checkarray);

    // Verify the FAXPY computatin at GPU is correct
    for (int i = 0; i < N; i++) {
      if (notEqual(checkarray[i], resultarray[i])) {
        fprintf(stderr, "Error: device axpy outputs incorrect result."
			" A[%d] = %.5f, expecting %.5f.\n", i, resultarray[i], checkarray[i]);
	exit(1);
      }
    }
    printf("device faxpy outputs are correct!\n");

    free(xarray);
    free(yarray);
    free(resultarray);
    free(checkarray);

    return 0;
}

